#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
typedef struct {
  int rows;
  int cols;
  int **mt;
} matrix;

int get_rows(int n) {
  printf("Enter rows of %d matrix\n", n);
  int r;
  scanf("%d", &r);
  return r;
}
int get_cols(int n) {
  printf("Enter cols of %d matrix\n", n);
  int r;
  scanf("%d", &r);
  return r;
}

void populate_matrix(matrix K) {
  for (int i = 0; i < K.rows; ++i) {
    for (int j = 0; j < K.cols; ++j) {
      K.mt[i][j] = rand() % 100;
    }
  }
}

void Print_matrix(matrix K) {
  for (int i = 0; i < K.rows; ++i) {
    for (int j = 0; j < K.cols; ++j) {
      printf("%d ", K.mt[i][j]);
    }
    printf("\n");
  }
}

int *linearize(matrix k) {
  int *L = (int *)malloc(k.rows * k.cols * sizeof(int));
  for (int i = 0; i < k.rows; i++) {
    for (int j = 0; j < k.cols; j++) {
      L[i * k.cols + j] = k.mt[i][j];
    }
  }
  return L;
}

__global__ void bleh_r(int *A, int *B, int rows, int cols) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  int stride = blockDim.x * gridDim.x;
  int size = rows * cols;
  for (int i = idx; i < size; i += stride) {
    B[i] = A[i] * 2;
  }
}
void time_taken(matrix A) {
  int *m_h = linearize(A);
  int *m_d, *o_d;
  hipMalloc(&m_d, A.rows * A.cols * sizeof(int));
  hipMalloc(&o_d, A.rows * A.cols * sizeof(int));
  hipMemcpy(m_d, m_h, A.rows * A.cols * sizeof(int), hipMemcpyHostToDevice);
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  // kerenl

  dim3 block(256);
  dim3 grid((A.rows * A.cols + block.x - 1) / block.x);

  bleh_r<<<grid, block>>>(m_d, o_d, A.rows, A.cols);

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float ms = 0;
  hipEventElapsedTime(&ms, start, stop);
  printf("The time taken to read atrix  is %f\n", ms);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipFree(m_d);
  hipFree(o_d);
  free(m_h);
}
int main() {
  matrix A;

  A.rows = get_rows(1);
  A.cols = get_cols(1);

  //-----------------------BUILDING A MATRIX--------------------------
  A.mt = (int **)malloc(A.rows * sizeof(int *));
  for (int i = 0; i < A.rows; ++i) {
    A.mt[i] = (int *)malloc(A.cols * sizeof(int));
  }

  // Populating the matrix
  populate_matrix(A);
  time_taken(A);
  for (int i = 0; i < A.rows; ++i) {
    free(A.mt[i]);
  }
  free(A.mt);
}
