// thread coarsening

#include <hip/hip_runtime.h>
#include <cstdlib>
#include <stdio.h>
#define SIZE 2048
#define COARSE_FACTOR 2

__global__ void sum_reduction_kernel_segmented_coarsed(int *input,
                                                       int *output) {
  __shared__ int in_s[SIZE / 2];
  unsigned int segment = COARSE_FACTOR * 2 * blockDim.x * blockIdx.x;
  unsigned int i = segment + threadIdx.x;
  unsigned int t = threadIdx.x; // ranges 0 2 4 6 8...........
  int sum = input[i];
  for (unsigned int tile = 1; tile < COARSE_FACTOR * 2; ++tile) {
    sum += input[i + tile * SIZE / 2];
  }
  in_s[t] = sum;
  __syncthreads();
  for (unsigned int stride = blockDim.x / 2; stride >= 1;
       stride /= 2) { // ranges as 1 2 4 8......
    if (t < stride) {
      in_s[t] += in_s[t + stride];
    }
    __syncthreads();
  }
  if (t == 0) {
    atomicAdd(output, in_s[0]);
  }
}
int main() {
  int *arr = (int *)malloc(sizeof(int) * SIZE);

  for (int i = 0; i < SIZE; ++i) {
    arr[i] = 1;
  }
  // printf("THE ARRAY IS \n");
  // for (int i = 0; i < SIZE; ++i) {
  //   printf("%d ", arr[i]);
  // }
  // printf("\n");
  //
  int output_h, *output_d;
  int *arr_d;

  hipMalloc(&arr_d, SIZE * sizeof(int));
  hipMalloc(&output_d, sizeof(int));
  hipMemset(&output_d, 0, sizeof(int));

  dim3 threads(SIZE / 2);

  hipMemcpy(arr_d, arr, SIZE * sizeof(int), hipMemcpyHostToDevice);

  sum_reduction_kernel_segmented_coarsed<<<(SIZE + threads.x - 1) / threads.x,
                                           threads>>>(arr_d, output_d);
  hipMemcpy(&output_h, output_d, sizeof(int), hipMemcpyDeviceToHost);

  printf("THE SUM OF ENTIRE ARRAY IS %d\n", output_h);
  return 0;
}
