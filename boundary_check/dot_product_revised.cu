#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
typedef struct {
  int rows;
  int cols;
  int **mt;
} matrix;

int get_rows(int n) {
  printf("Enter rows of %d matrix\n", n);
  int r;
  scanf("%d", &r);
  return r;
}
int get_cols(int n) {
  printf("Enter cols of %d matrix\n", n);
  int r;
  scanf("%d", &r);
  return r;
}

void populate_matrix(matrix K) {
  for (int i = 0; i < K.rows; ++i) {
    for (int j = 0; j < K.cols; ++j) {
      K.mt[i][j] = rand() % 100;
    }
  }
}

void Print_matrix(matrix K) {
  for (int i = 0; i < K.rows; ++i) {
    for (int j = 0; j < K.cols; ++j) {
      printf("%d ", K.mt[i][j]);
    }
    printf("\n");
  }
}

int *linearize(matrix k) {
  int *L = (int *)malloc(k.rows * k.cols * sizeof(int));
  for (int i = 0; i < k.rows; i++) {
    for (int j = 0; j < k.cols; j++) {
      L[i * k.cols + j] = k.mt[i][j];
    }
  }
  return L;
}

// delinearize the matrix
void delinearize(matrix K, int *l) {

  for (int i = 0; i < K.rows; ++i) {
    for (int j = 0; j < K.cols; ++j) {
      K.mt[i][j] = l[i * K.cols + j];
    }
  }
}
#define TILE_WIDTH 2

__global__ void dotKerenl(int *A, int *B, int *C, int A_rows, int A_cols,
                          int B_cols) {

  __shared__ int Mds[TILE_WIDTH][TILE_WIDTH];
  __shared__ int Nds[TILE_WIDTH][TILE_WIDTH];

  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int row = by * TILE_WIDTH + ty;
  int col = bx * TILE_WIDTH + tx;
  int Pval = 0;
  for (int i = 0; i < (A_cols + TILE_WIDTH - 1) / TILE_WIDTH; ++i) {
    if ((row < A_rows) && (i * TILE_WIDTH + tx) < A_cols) {
      Mds[ty][tx] = A[row * A_cols + i * TILE_WIDTH + tx];
    } else {
      Mds[ty][tx] = 0;
    }
    if ((i * TILE_WIDTH + ty) < A_cols && col < B_cols) {
      Nds[ty][tx] = B[(i * TILE_WIDTH + ty) * B_cols + col];
    } else {
      Nds[ty][tx] = 0;
    }
    __syncthreads();
    for (int k = 0; k < TILE_WIDTH; ++k) {
      Pval += Mds[ty][k] * Nds[k][tx];
    }
    __syncthreads();
  }
  if ((row < A_rows) && (col < B_cols)) {
    C[row * B_cols + col] = Pval;
  }
}
void matrixMulKerenel(matrix A, matrix B, matrix C) {
  // get linear versions
  int *A_l = linearize(A), *B_l = linearize(B),
      *C_l = (int *)malloc(C.rows * C.cols * sizeof(int));
  ;
  int *A_d, *B_d, *C_d;
  hipMalloc((void **)&A_d, A.rows * A.cols * sizeof(int));
  hipMalloc((void **)&B_d, B.rows * B.cols * sizeof(int));
  hipMalloc((void **)&C_d, C.rows * C.cols * sizeof(int));

  hipMemcpy(A_d, A_l, A.rows * A.cols * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(B_d, B_l, B.rows * B.cols * sizeof(int), hipMemcpyHostToDevice);

  dim3 Block(TILE_WIDTH, TILE_WIDTH);
  dim3 Grid((C.cols + TILE_WIDTH - 1) / TILE_WIDTH,
            (C.rows + TILE_WIDTH - 1) / TILE_WIDTH);

  // kernel call
  dotKerenl<<<Grid, Block>>>(A_d, B_d, C_d, A.rows, A.cols, B.cols);
  hipMemcpy(C_l, C_d, C.rows * C.cols * sizeof(int), hipMemcpyDeviceToHost);
  free(A_l);
  free(B_l);
  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);
  delinearize(C, C_l);
  free(C_l);
}

int main() {
  matrix A;
  matrix B;

  A.rows = get_rows(1);
  A.cols = get_cols(1);
  B.rows = get_rows(2);
  B.cols = get_cols(2);

  //-----------------------BUILDING A MATRIX--------------------------
  A.mt = (int **)malloc(A.rows * sizeof(int *));
  for (int i = 0; i < A.rows; ++i) {
    A.mt[i] = (int *)malloc(A.cols * sizeof(int));
  }
  B.mt = (int **)malloc(B.rows * sizeof(int *));
  for (int i = 0; i < B.rows; ++i) {
    B.mt[i] = (int *)malloc(B.cols * sizeof(int));
  }

  // Populating the matrix
  populate_matrix(A);
  populate_matrix(B);

  if (A.cols != B.rows) {
    printf("No proper dimensions exiting...............\n");

    for (int i = 0; i < A.rows; ++i)
      free(A.mt[i]);
    free(A.mt);
    for (int i = 0; i < B.rows; ++i)
      free(B.mt[i]);
    free(B.mt);

    exit(0);
  } else {
    matrix C;
    C.rows = A.rows;
    C.cols = B.cols;
    C.mt = (int **)malloc(C.rows * sizeof(int *));
    for (int i = 0; i < C.rows; ++i) {
      C.mt[i] = (int *)malloc(C.cols * sizeof(int));
    }
    printf("----------MATRIX A---------------\n");
    Print_matrix(A);
    printf("----------MATRIX B---------------\n");
    Print_matrix(B);
    printf(
        "--------------------MULTIPLYING THE MATRICES---------------------\n");
    matrixMulKerenel(A, B, C);

    Print_matrix(C);
    for (int i = 0; i < C.rows; ++i) {
      free(C.mt[i]);
    }
    free(C.mt);
  }
}
