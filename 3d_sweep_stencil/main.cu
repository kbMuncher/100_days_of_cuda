#include <hip/hip_runtime.h>
#include <iostream>

#define N 8
#define BLOCK_SIZE 4

__constant__ float c0 = 0.01f;
__constant__ float c1 = 0.05f;
__constant__ float c2 = 0.09f;
__constant__ float c3 = 0.01f;
__constant__ float c4 = 0.09f;
__constant__ float c5 = 0.05f;
__constant__ float c6 = 0.01f;

__global__ void stencil_sweep_3D(float *in, float *out, int n) {
  int i = blockDim.z * blockIdx.z + threadIdx.z; // [N, N , _]
  int j = blockIdx.y * blockDim.y + threadIdx.y; //[N ,_, N]
  int k = blockIdx.x * blockDim.x + threadIdx.x; //[_,N,N]

  if (i > 0 && i < n - 1 && j > 0 && j < n - 1 && k > 0 && k < n - 1) {
    out[i * n * n + j * n + k] = c0 * in[i * n * n + j * n + k] +
                                 c1 * in[(i - 1) * n * n + j * n + k] +
                                 c2 * in[(i + 1) * n * n + j * n + k] +
                                 c3 * in[i * n * n + (j - 1) * n + k] +
                                 c4 * in[i * n * n + (j + 1) * n + k] +
                                 c5 * in[i * n * n + j * n + (k - 1)] +
                                 c6 * in[i * n * n + j * n + (k + 1)];
  }
}

int main() {
  unsigned int size = N * N * N * sizeof(float);
  float *in_h = new float[N * N * N];
  float *out_h = new float[N * N * N];

  // populate the matrix
  for (int i = 0; i < N; ++i) {
    for (int j = 0; j < N; ++j) {
      for (int k = 0; k < N; ++k) {
        in_h[i * N * N + j * N + k] = static_cast<float>(i * N * N + j * N + k);
      }
    }
  }
  // kernel config
  float *in_d, *out_d;
  hipMalloc(&in_d, size);
  hipMalloc(&out_d, size);

  hipMemcpy(in_d, in_h, size, hipMemcpyHostToDevice);
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
  dim3 grid(2, 2,
            2); // use (BLOCK_SIZE+N-1)/BLOCK_SIZE
                // now this means tioal threads per block is 4*4*4 = 64 amd
                // blocks per grid is 8 i.e thread per kernel launch is 512

  stencil_sweep_3D<<<grid, block>>>(in_d, out_d, N);

  hipMemcpy(out_h, out_d, size, hipMemcpyDeviceToHost);
  hipFree(in_d);
  hipFree(out_d);

  printf("____________________________________________________\n");
  printf("FIRST A SLICE FROM ORIGIANL MATRIX (lets take FIRST slcie\n");
  for (int j = 0; j < N; ++j) {
    for (int k = 0; k < N; ++k) {
      printf("%.2f ", in_h[1 * N * N + j * N + k]);
    }
    printf("\n");
  }
  printf("____________________________________________________\n");
  printf("NOW CORRESPONDING SLICE FROM THE OUTPUT \n");
  for (int j = 0; j < N; ++j) {
    for (int k = 0; k < N; ++k) {
      printf("%.2f ", out_h[1 * N * N + j * N + k]);
    }
    printf("\n");
  }
  delete[] in_h;
  delete[] out_h;
  return 0;
}
